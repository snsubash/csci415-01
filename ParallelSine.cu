#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <sys/time.h>

// problem size (vector length) N
static const int N = 12345678;

// Number of terms to use when approximating sine
static const int TERMS = 6;

// kernel function (CPU - Do not modify)
void sine_serial(float *input, float *output)
{
  int i;

  for (i=0; i<N; i++) {
      float value = input[i]; 
      float numer = input[i] * input[i] * input[i]; 
      int denom = 6; // 3! 
      int sign = -1; 
      for (int j=1; j<=TERMS;j++) 
      { 
         value += sign * numer / denom; 
         numer *= input[i] * input[i]; 
         denom *= (2*j+2) * (2*j+3); 
         sign *= -1; 
      } 
      output[i] = value; 
    }
}


// kernel function (CUDA device)
// TODO: Implement your graphics kernel here. See assignment instructions for method information

__global__ void sine_parallel(float *input, float *output)
{
	// dimension structure with fields
  int idx = blockIdx.x * 512 + threadIdx.x;
   
  float value = input[idx];
	// multiplying by 3
  float numer = input[idx] * input[idx] * input[idx];
  int denom = 6;
  int sign = -1;
  for (int i = 1; i<=TERMS; i++)
  {
	value += sign * numer / denom;
	numer *= input[idx] * input[idx];
	denom *= (2 * i + 2) * (2 * i + 3);
	sign *= -1;
  }
	// result writing into output array
  output[idx] = value;
}

// BEGIN: timing and error checking routines (do not modify)

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}

void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

// END: timing and error checking routines (do not modify)



int main (int argc, char **argv)
{
  //BEGIN: CPU implementation (do not modify)
  float *h_cpu_result = (float*)malloc(N*sizeof(float));
  float *h_input = (float*)malloc(N*sizeof(float));
  //Initialize data on CPU
  int i;
  for (i=0; i<N; i++)
  {
    h_input[i] = 0.1f * i;
  }

  //Execute and time the CPU version
  long long CPU_start_time = start_timer();
  sine_serial(h_input, h_cpu_result);
  long long CPU_time = stop_timer(CPU_start_time, "\nCPU Run Time");
  //END: CPU implementation (do not modify)


  //TODO: Prepare and run your kernel, make sure to copy your results back into h_gpu_result and display your timing results
 
// instantiating constat values
  const int sizeOfBlock = 512;
  const int sizeOfGrid = N/512 + 1; 
  const float bytes = N * sizeof(float);
	
//Declaring the arrays  
  float *d_input;
  float *d_output;

	
  // timer for GPU start time
  long long GPU_startTotal = start_timer();
   
  //Allocating memory
  float *h_gpu_result = (float*)malloc(bytes);
  
  //Allocating memory to the GPU and timing it
  long long GPU_allocateStart = start_timer();
  hipMalloc((void**) &d_input, bytes);
  hipMalloc((void**) &d_output, bytes);
  long long GPU_allocateTime = stop_timer(GPU_allocateStart, "\nGPU Memory Allocation");

  // Copying input to output and timing it
  long long GPU_dcopyStart = start_timer();
  hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);
  long long GPU_dcopyTime = stop_timer(GPU_dcopyStart, "Copying GPU Memory to Device");

  //Launching the kernel with N threads and then timing it
  long long GPU_kernelStart = start_timer();
  sine_parallel<<<sizeOfGrid, sizeOfBlock>>>(d_input, d_output);
  long long GPU_kernelTime = stop_timer(GPU_kernelStart, "GPU Kernel Run Time");

  //Copying the output of parallel_sine back to the host (h_gpu_result) and timing it
  long long GPU_hcopyStart = start_timer();
  hipMemcpy(h_gpu_result, d_output, bytes, hipMemcpyDeviceToHost);
  long long GPU_hcopyTime = stop_timer(GPU_hcopyStart, "Copying GPU Memory to Host");
	
  
  //Free GPU memory
  hipFree(d_input);
  hipFree(d_output);

  // End timer for GPU
  long long GPU_totalTime = stop_timer(GPU_startTotal, "Total GPU Run Time");

  // Checking to make sure the CPU and GPU results match - Do not modify
  int errorCount = 0;
  for (i=0; i<N; i++)
  {
    if (abs(h_cpu_result[i]-h_gpu_result[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
  else
    printf("Result comparison passed.\n");

  // Cleaning up memory
  free(h_input);
  free(h_cpu_result);
  free(h_gpu_result);
  return 0;
}
